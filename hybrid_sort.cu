#include <iostream>
#include <fstream>
#include <xmmintrin.h>
#include <hip/hip_runtime.h>
#include <omp.h>
#include <boost/timer/timer.hpp>
#include <boost/format.hpp>
#include <test/test_util.h>
#include <cub/util_allocator.cuh>
#include <cub/device/device_radix_sort.cuh>
#include <cub/util_type.cuh>
#include "util.h"
#include "cpu_sort.h"

const int cacheFactor = 4; //what is the most suitable cache size?
const size_t chunkFactor = 1;
const size_t dlFactor = 27;

float gpu_sort(float *data, size_t dataLen, size_t blockLen);
void gpu_sort_serial(float *data, size_t dataLen, size_t blockLen);
void gpu_sort_test(float *data, rsize_t dataLen);
float *cpu_sort_sse_parallel(DoubleBuffer<float> &data, rsize_t dataLen);
void hybrid_sort(float *data, size_t dataLen);

int main(int argc, char **argv)
{
	rsize_t dataLen = (1 << dlFactor) * chunkFactor; //default length of sorted data
	int seed = 1023;  //default seed for generate random data sequence
	//std::cout << omp_get_max_threads() << std::endl;
	CommandLineArgs args(argc, argv);
	args.GetCmdLineArgument("l", dataLen);
	args.GetCmdLineArgument("s", seed);
	//std::cout << dataLen << " " << seed << "\n";
	args.DeviceInit();
	/*float *data = new float[dataLen];
	GenerateData(seed, data, dataLen);
	gpu_sort_test(data, dataLen);
	gpu_sort_serial(data, dataLen, dataLen);
	delete [] data;*/
	//hipSetDeviceFlags(hipDeviceScheduleBlockingSync);
	for (int dlf = 20; dlf <= 21; ++dlf)
	{
		dataLen = 1 << dlf;
		std::cout << "data length: " << dataLen << std::endl;
		float *data = new float[dataLen];
		GenerateData(seed, data, dataLen);
		hybrid_sort(data, dataLen);
		delete [] data;
		//std::cout << "loop time: " << dlf << std::endl;
	}
	std::cout << "test complete." << std::endl;
	//resultTest(cpu_sort_sse_parallel(hdata, dataLen), dataLen);
	//resultTest(mergeSortInBlockParallel(dataIn, dataOut, dataLen), dataLen);
	//gpu_sort(dataIn, dataLen, dataLen >> 2);
	//gpu_sort_serial(dataIn, dataLen, dataLen >>2);
		/*#pragma omp parallel
		  {
		omp_set_nested(1);
#pragma omp single nowait
std::cout << "single run" << omp_get_nested() << std::endl;
		gpu_sort(data, dataLen);
		#pragma omp single
		  resultTest(data, dataLen);
		  #pragma omp parallel
		std::cout << omp_get_thread_num();
		}*/
	return 0;
}

//using stream to overlap kernal excution and data transfer between CPU and GPU.
//all sorting task broken to 2 parts, the first will overlap data upload to GPU,
//the second will overlap data download from CPU.
float gpu_sort(float *data, size_t dataLen, size_t blockLen)
{
	//boost::timer::auto_cpu_timer t;
	int blockNum = dataLen / blockLen;
	size_t blockBytes = sizeof(float) * blockLen;
	hipStream_t *streams = new hipStream_t[blockNum];
	for (int i = 0; i < blockNum; ++i)
		hipStreamCreate(&streams[i]);
    hipcub::DoubleBuffer<float> d_keys;
    hipcub::CachingDeviceAllocator cda;
    cda.DeviceAllocate((void**) &d_keys.d_buffers[0], sizeof(float) * dataLen);
    cda.DeviceAllocate((void**) &d_keys.d_buffers[1], sizeof(float) * dataLen);
    void *d_temp_storage = NULL;
    size_t temp_storage_bytes = 0;
	hipcub::DeviceRadixSort::SortKeys(d_temp_storage, temp_storage_bytes, d_keys,
								   blockLen);
	cda.DeviceAllocate(&d_temp_storage, temp_storage_bytes);
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	hipMemcpyAsync(d_keys.d_buffers[d_keys.selector], data, blockBytes, hipMemcpyHostToDevice, streams[0]);
	int remain_to_upload = blockNum - 1;
	int upload_loop = remain_to_upload >> 1;
	size_t offset = 0;
	size_t up_offset = blockLen;
	for (int i = 0; i < upload_loop; ++i)
	{
		hipcub::DoubleBuffer<float> chunk(d_keys.d_buffers[d_keys.selector] + offset, d_keys.d_buffers[d_keys.selector ^ 1] + offset);
		hipcub::DeviceRadixSort::SortKeys(d_temp_storage, temp_storage_bytes, chunk, blockLen, 0, 32, streams[i]);
		int upload_blocks = 2 + remain_to_upload % 2;
		hipMemcpyAsync(d_keys.d_buffers[d_keys.selector] + up_offset, data + up_offset, upload_blocks * blockBytes, hipMemcpyHostToDevice, streams[i + 1]);
		remain_to_upload -= upload_blocks;
		up_offset += upload_blocks * blockLen;
		offset += blockLen;
	}
	int selector = d_keys.selector ^ 1;
	int remain_to_donwload = upload_loop;
	size_t down_offset = 0;
	for (int i = upload_loop; i < blockNum; ++i)
	{
		hipcub::DoubleBuffer<float> chunk(d_keys.d_buffers[d_keys.selector] + offset, d_keys.d_buffers[d_keys.selector ^ 1] + offset);
		hipcub::DeviceRadixSort::SortKeys(d_temp_storage, temp_storage_bytes, chunk, blockLen, 0, 32, streams[i]);
		int dowload_blocks = 1 + (remain_to_donwload > 1);
		hipMemcpyAsync(data + down_offset, d_keys.d_buffers[selector] + down_offset, dowload_blocks * blockBytes, hipMemcpyDeviceToHost, streams[i - 1]);
		remain_to_donwload -= (dowload_blocks - 1);
		down_offset += dowload_blocks * blockLen;
		offset += blockLen;
	}
	hipMemcpyAsync(data + dataLen - blockLen, d_keys.d_buffers[selector] + dataLen - blockLen, sizeof(float) * blockLen, hipMemcpyDeviceToHost, streams[blockNum - 1]);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	float sort_time;
	hipEventElapsedTime(&sort_time, start, stop);
	/*std::cout << "time used on gpu sort loop: " << sort_time << std::endl;
	for (size_t offset = 0; offset < dataLen; offset += blockLen)
	resultTest(data + offset, blockLen);*/
	for (int i = 0; i < blockNum; ++i)
		hipStreamDestroy(streams[i]);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	cda.DeviceFree(d_keys.d_buffers[0]);
	cda.DeviceFree(d_keys.d_buffers[1]);
	cda.DeviceFree(d_temp_storage);
	return sort_time;
}

void gpu_sort_serial(float *data, size_t dataLen, size_t blockLen)
{
	boost::timer::auto_cpu_timer t;
    hipcub::DoubleBuffer<float> d_keys;
    hipcub::CachingDeviceAllocator cda;
    cda.DeviceAllocate((void**) &d_keys.d_buffers[0], sizeof(float) * dataLen);
    cda.DeviceAllocate((void**) &d_keys.d_buffers[1], sizeof(float) * dataLen);
    void *d_temp_storage = NULL;
    size_t temp_storage_bytes = 0;
	hipcub::DeviceRadixSort::SortKeys(d_temp_storage, temp_storage_bytes, d_keys,
								   blockLen);
	cda.DeviceAllocate(&d_temp_storage, temp_storage_bytes);
	hipEvent_t start, stop;
	float sort_time = 0, transfer_time;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, 0);
	hipMemcpyAsync(d_keys.d_buffers[d_keys.selector], data, sizeof(float) * dataLen, hipMemcpyHostToDevice);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&transfer_time, start, stop);
	std::cout << "time used for host to device transfer: " << transfer_time << std::endl;
	hipMemsetAsync(d_keys.d_buffers[d_keys.selector ^ 1], 0, sizeof(float) * dataLen);
	for (size_t offset = 0; offset < dataLen; offset += blockLen)
	{
		float stime;
		hipEventRecord(start, 0);
	    hipcub::DoubleBuffer<float> chunk(d_keys.d_buffers[d_keys.selector] + offset, d_keys.d_buffers[d_keys.selector ^ 1] + offset);
		hipcub::DeviceRadixSort::SortKeys(d_temp_storage, temp_storage_bytes, chunk, blockLen);
		hipEventRecord(stop, 0);
		hipEventSynchronize(stop);
		hipEventElapsedTime(&stime, start, stop);
		sort_time += stime;
		//d_keys.selector = chunk.selector;
	}
	std::cout << "average time used for block sort:" << sort_time * (blockLen * 1.0 / dataLen) << std::endl;
	d_keys.selector ^= 1;
	hipEventRecord(start, 0);
	hipMemcpyAsync(data, d_keys.Current(), sizeof(float) * dataLen, hipMemcpyDeviceToHost);
	hipEventRecord(stop, 0);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&transfer_time, start, stop);
	std::cout << "time used for device to host transfer: " << transfer_time << std::endl;
	for (size_t offset = 0; offset < dataLen; offset += blockLen)
		resultTest(data + offset, blockLen);
	cda.DeviceFree(d_keys.d_buffers[0]);
	cda.DeviceFree(d_keys.d_buffers[1]);
	cda.DeviceFree(d_temp_storage);
}

void gpu_sort_test(float *data, rsize_t dataLen)
{
	if (dataLen < (1 << 20) || dataLen > (1 << 28))
	{
		std::cout << "data length too short or too long!" << std::endl;
		return;
	}
	std::ofstream rFile("/home/aloneranger/source_code/Hybrid_Sort/result.txt",
						std::ios::app);
	rFile << "gpu kernel and transfer test" << std::endl
		  << boost::format("%1%%|15t|") % "data length"
		  << boost::format("%1%%|15t|") % "transfer time"
		  << boost::format("%1%%|15t|") % "kernel time"
		  << std::endl;
	hipcub::DoubleBuffer<float> d_keys;
    hipcub::CachingDeviceAllocator cda;
    cda.DeviceAllocate((void**)&d_keys.d_buffers[0], sizeof(float) * dataLen);
    cda.DeviceAllocate((void**)&d_keys.d_buffers[1], sizeof(float) * dataLen);
	hipEvent_t start, stop;
	hipEventCreate(&start);
	hipEventCreate(&stop);
	int test_time = 50;
	for (size_t chunk_size = 1 << 17; chunk_size <= dataLen; chunk_size *= 2)
	{
		void *d_temp_storage = NULL;
		size_t temp_storage_bytes = 0;
		hipcub::DeviceRadixSort::SortKeys(d_temp_storage, temp_storage_bytes,
									   d_keys, chunk_size);
		cda.DeviceAllocate(&d_temp_storage, temp_storage_bytes);
		float transfer_time = 0.0, kernel_time = 0.0;
		size_t offset = 0;
		for (int i = 0; i < test_time; ++i)
		{
			if (offset == dataLen) offset = 0;
			hipEventRecord(start, 0);
			hipMemcpy(d_keys.d_buffers[0] + offset, data + offset,
					   sizeof(float) * chunk_size, hipMemcpyHostToDevice);
			hipEventRecord(stop, 0);
			hipEventSynchronize(stop);
			float ttime;
			hipEventElapsedTime(&ttime, start, stop);
			transfer_time += ttime;
			hipcub::DoubleBuffer<float> chunk(d_keys.d_buffers[0] + offset,
										   d_keys.d_buffers[1] + offset);
			hipEventRecord(start, 0);
			hipcub::DeviceRadixSort::SortKeys(d_temp_storage, temp_storage_bytes,
										   chunk, chunk_size);
			hipEventRecord(stop, 0);
			hipEventSynchronize(stop);
			float ktime;
			hipEventElapsedTime(&ktime, start, stop);
			kernel_time += ktime;
			offset += chunk_size;
		}
		rFile << boost::format("%1%%|15t|") % chunk_size
			  << boost::format("%1%%|15t|") % (transfer_time / test_time)
			  << boost::format("%1%%|15t|") % (kernel_time / test_time)
			  << std::endl;
		cda.DeviceFree(d_temp_storage);
	}
	
    /*hipMemset(d_keys.d_buffers[d_keys.selector ^ 1], 0,
	  sizeof(float) * dataLen);*/
    /*hipMemcpy(data, d_keys.Current(), sizeof(float) * dataLen,
	  hipMemcpyDeviceToHost);*/
	cda.DeviceFree(d_keys.d_buffers[0]);
	cda.DeviceFree(d_keys.d_buffers[1]);
	rFile << std::endl << std::endl;
	rFile.close();
}

float *cpu_sort_sse_parallel(DoubleBuffer<float> &data, rsize_t dataLen)
{
	boost::timer::auto_cpu_timer t;
	const rsize_t blockSize = cacheSizeInByte() / (cacheFactor * sizeof(float));
    std::cout << "selected block size: " << blockSize << std::endl;
	if (blockSize)
	{
		int w, selector = 0;
		updateMergeSelcetor(&selector, blockSize);
		//if serialized, then this single code block is enough
#pragma omp parallel private(w) firstprivate(selector)
		{
			int threads = omp_get_num_threads();
			w = omp_get_thread_num();
			rsize_t chunkSize = dataLen / threads;
			rsize_t chunkStart = w * chunkSize;
			for (rsize_t offset = chunkStart; offset < chunkStart + chunkSize;
				 offset += blockSize)
			{
				DoubleBuffer<float> chunk(data.buffers[data.selector] + offset, data.buffers[data.selector ^ 1] + offset);
				mergeSort(chunk, blockSize);
			}
			DoubleBuffer<float> chunk(data.buffers[selector] + chunkStart,
									  data.buffers[selector ^ 1] + chunkStart);
			//chunk.selector = selector ^ data.selector;
			registerSortIteration(chunk, blockSize * 2, chunkSize, chunkSize);
#pragma omp single
			data.selector = chunk.selector;
		}
		
		rsize_t blockNum = (dataLen / blockSize);
#pragma omp parallel private(w)
		{
			int threads = omp_get_num_threads();
			rsize_t blocksPerThread = blockNum /threads;
			w = omp_get_thread_num();
			rsize_t quantileStart = w * blocksPerThread;
			rsize_t chunkSize = blockSize * blocksPerThread;
			rsize_t chunkStart = quantileStart * blockSize;
			multiWayMerge(data, dataLen, chunkSize, blockSize, quantileStart,
						  quantileStart + blocksPerThread);
			//data.selector ^= 1;	 
			for (rsize_t offset = chunkStart; offset < chunkStart + chunkSize;
				 offset += blockSize)
			{
				DoubleBuffer<float> chunk(data.buffers[data.selector] + offset,
										  data.buffers[data.selector ^ 1] + offset);
				chunk.selector ^= 1;
				mergeSort(chunk, blockSize);
			}
		}
		updateMergeSelcetor(&data.selector, blockSize);
		data.selector ^= 1;
	}
	return data.Current();
}

void mergeStage(DoubleBuffer<float> &data, size_t dataLen, size_t chunkSize,
				size_t blockSize)
{
	//boost::timer::auto_cpu_timer t;
	//updateMergeSelcetor(&data.selector, blockSize);
	//data.selector ^= 1;
	updateSelectorGeneral(data.selector, blockSize);
#pragma omp parallel 
	{
#pragma omp for 
		for (size_t cOffset = 0; cOffset < dataLen; cOffset += chunkSize)
		{
			for (size_t bOffset = cOffset; bOffset < cOffset + chunkSize;
				 bOffset += blockSize)
			{
				DoubleBuffer<float> block(data.buffers[0] + bOffset,
										  data.buffers[1] + bOffset);
				//mergeSort(block, blockSize);
				mergeSortGeneral(block, blockSize);
			}
			DoubleBuffer<float> chunk(data.buffers[data.selector] + cOffset,
									  data.buffers[data.selector ^ 1] + cOffset);
			registerSortIteration(chunk, blockSize * 2, chunkSize, chunkSize);
		}
	}
	updateMergeSelcetor(&data.selector, chunkSize);
}

void multiWayStage(DoubleBuffer<float> &data, size_t dataLen, size_t chunkSize,
				   size_t blockSize)
{
	//boost::timer::auto_cpu_timer t;
#pragma omp parallel
	{
#pragma omp for
		for (size_t cOffset = 0; cOffset < dataLen; cOffset += chunkSize)
		{
			multiWayMergeGeneral(data, dataLen, chunkSize, blockSize,
								 cOffset, cOffset + chunkSize);
		}
	}
	data.selector ^= 1;
#pragma omp parallel
	{
#pragma omp for
		for (size_t cOffset = 0; cOffset < dataLen; cOffset += chunkSize)
			for (size_t boffset = cOffset; boffset < cOffset + chunkSize;
				 boffset += blockSize)
			{
				DoubleBuffer<float> block(data.buffers[data.selector] + boffset,
										  data.buffers[data.selector ^ 1] + boffset);
				//mergeSort(block, blockSize);
				mergeSortGeneral(block, blockSize);
			}
	}
	//updateMergeSelcetor(&data.selector, blockSize);
	//data.selector ^= 1;
	updateSelectorGeneral(data.selector, blockSize);
}

void hybrid_sort(float *data, size_t dataLen)
{
	std::ofstream rFile("/home/aloneranger/source_code/Hybrid_Sort/result.txt",
						std::ios::app);
	if (rFile.is_open()) rFile << "tested data length: " << dataLen << std::endl;
	float* dataIn = (float*)_mm_malloc(dataLen * sizeof(float), 16);
	float* dataOut= (float*)_mm_malloc(dataLen * sizeof(float), 16);
	std::copy(data, data + dataLen, dataIn);
	DoubleBuffer<float> hdata(dataIn, dataOut);
	const size_t chunkSize = dataLen / chunkFactor;
	const size_t blockSize = cacheSizeInByte() / (cacheFactor * sizeof(float));
	std::cout << "selected block size on cpu: " << blockSize << std::endl;
	//TODO: if blockSize == 0, give it a default value.
	mergeStage(hdata, dataLen, chunkSize, blockSize);
	multiWayStage(hdata, dataLen, chunkSize, blockSize);
	resultTest(hdata.Current(), dataLen);
	/*std::copy(data, data + dataLen, dataIn);
	gpu_sort(dataIn, dataLen, dataLen / 8);
	hipDeviceSynchronize();*/
	
	const int test_time = 50;
	rFile << boost::format("%1%%|15t|") % "cache factor"
		  << boost::format("%1%%|15t|") % "block length"
		  << boost::format("%1%%|15t|") % "chunk size"
		  << boost::format("%1%%|15t|") % "merge time"
		  << boost::format("%1%%|15t|") % "multi way"
		//<< boost::format("%1%%|15t|") % "gpu omp"
		//<< boost::format("%1%%|15t|") % "gpu cuda"
		  << std::endl;
	for (int j = 1; j <= 64; j *= 2)
	{
		size_t block_size = cacheSizeInByte() / (j * sizeof(float));
		for (int m = 8; m <= 64; m *= 2)
		{
			double merge_time = 0.0, multiway_time = 0.0; //gpu_time = 0.0;
			//float cuda_time = 0.0;
			size_t chunk_size = dataLen / m;
			for (int i = 0; i < test_time; ++i)
			{
				double start, end;
				/*std::copy(data, data + dataLen, dataIn);
				start = omp_get_wtime();
				cuda_time += gpu_sort(dataIn, dataLen, chunk_size);
				hipDeviceSynchronize();
				end = omp_get_wtime();
				gpu_time += (end - start);*/
				std::copy(data, data + dataLen, dataIn);
				hdata.selector = 0;
				start = omp_get_wtime();
				mergeStage(hdata, dataLen, chunk_size, block_size);
				end = omp_get_wtime();
				merge_time += (end - start);
				start = omp_get_wtime();
				multiWayStage(hdata, dataLen, chunk_size, block_size);
				end = omp_get_wtime();
				multiway_time += (end - start);
			}
			rFile << boost::format("%1%%|15t|") % j
				  << boost::format("%1%%|15t|") % block_size
				  << boost::format("%1%%|15t|") % chunk_size
				  << boost::format("%1%%|15t|") % (merge_time / test_time)
				  << boost::format("%1%%|15t|") % (multiway_time / test_time)
				//<< boost::format("%1%%|15t|") % (gpu_time / test_time)
				//<< boost::format("%1%%|15t|") % (cuda_time / test_time)
				  << std::endl;
		}
	}
	_mm_free(dataIn);
	_mm_free(dataOut);
	rFile << std::endl << std::endl;
	rFile.close();
}

